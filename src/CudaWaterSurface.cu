#include "hip/hip_runtime.h"
/* 
 * File: CudaSurface.cu
 *
 * Copyright (C) 2019  Camille Schreck
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <https://www.gnu.org/licenses/>.
 */

#include "CudaWaterSurface.hpp"
#include "settings.hpp"
#include "definitions.hpp"
#include <iostream>
#include "error.hpp"
#include <boost/math/special_functions/bessel.hpp>
#include <hip/hip_fp16.h>
#include "ui_parameters.hpp"

using namespace settings;
using namespace definitions;
    
__global__
void addHeight(FLOAT *heights, FLOAT *displacement, FLOAT *amplitudes, FLOAT *indexes,
	       FLOAT *positions, bool *is_active, FLOAT *positions_grid, FLOAT *sizes,
	       uint nb_sources, uint nb_sources_input,
	       FLOAT k, FLOAT omega, FLOAT vel,
	       FLOAT time, FLOAT dt, uint nb_rows, uint nb_cols,
	       int size_ampli, int ampli_step, FLOAT damping, FLOAT scale,
	       bool show_input, bool show_scattered,
	       FLOAT *hankel_r_tab, FLOAT *hankel_i_tab ) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (uint i = index; i < nb_rows*nb_cols; i += stride) {
#ifdef PROJECTED_GRID
    FLOAT wl = 2*M_PI/k;
    if (sizes[i] >= wl/2.0) {
      return;
    }
      // position of the projected grid are recorded with the viewer coordinates
      // we need to set them in world coordinates
      FLOAT x = (1+positions_grid[2*i])*scale/2.0;
      FLOAT y = (1+positions_grid[2*i+1])*scale/2.0;
  
      // FLOAT dispxr = 0, dispyr = 0;
      // FLOAT dispxi = 0, dispyi = 0;
      FLOAT mod = 1;
      if (sizes[i] > wl/4.0) {
	mod = 1 - (sizes[i] - wl/4.0)/(wl/4.0);
      }
#else
    FLOAT x = scale/(FLOAT)nb_rows*(i/nb_cols);
    FLOAT y = scale/(FLOAT)nb_cols*(i%nb_cols);
    FLOAT mod = 1;
#endif
    FLOAT hr = 0, hi = 0;
      uint start = 0, end = nb_sources + nb_sources_input;
      if (!show_input) {
	start = nb_sources_input;
      }
      if (!show_scattered) {
	end = nb_sources_input;
      }

      for (uint s = start; s < end; ++s) {
      while (!is_active[s] && s < end) {
	++s;
      }
      if (s >= end) {
	break;
      }
      if (indexes[s] == 1) {
	if (time > x/vel) {
	  FLOAT dx = positions[2*s];
	  FLOAT dy = positions[2*s + 1];
	  FLOAT kx = k*(x*dx + y*dy);
	  FLOAT ar = amplitudes[2*s*size_ampli];
	  FLOAT ai = amplitudes[2*s*size_ampli + 1];
	  hr += ar*cos(kx) - ai*sin(kx);
	  hi += ai*cos(kx) + ar*sin(kx);
	  // dispxr -= hi*dx;
	  // dispxi += hr*dx;
	  // dispyr -= hi*dy;
	  // dispyi += hr*dy;
	}
      } else {
	FLOAT rx = x - positions[2*s];
	FLOAT ry = y - positions[2*s+1];
	  
	FLOAT r = sqrt((float)(rx*rx + ry*ry));
	// FLOAT kx = rx/r;
	// FLOAT ky = ry/r;
	FLOAT damp = exp(-damping*k*k*r);
	if (damp > 0.02) { 
	  FLOAT ret = r/vel;
	  int l = floor((time-ret)/((FLOAT)ampli_step*dt));
	  if (time-ret < 0) {
	    --l;
	  }

	  FLOAT ar = 0, ai = 0;
	  // if (l > size_ampli-1) {
	  //   // ar = amplitudes[2*(s*size_ampli + size_ampli-1)];
	  //   // ai = amplitudes[2*(s*size_ampli + size_ampli-1)+1];
	  // } else
	    if (l < 0) {
	    ar = amplitudes[2*s*size_ampli];
	    ai = amplitudes[2*s*size_ampli+1];
	  
	  } else {
	    FLOAT t = time-ret;
	    FLOAT w = 0;
	    FLOAT tl = l*ampli_step*dt;
	    FLOAT tl_prev = (l-1)*ampli_step*dt, tl_next = (l+1)*ampli_step*dt;
	    if (t < tl_prev || t > tl_next) {
	      w = 0;
	    } else if (t < tl) {
	      w = (t - tl_prev)/(dt*(FLOAT)ampli_step);
	    } else {
	      w = (tl_next - t)/(dt*(FLOAT)ampli_step);
	    }
	    //	    w = 1;
	    ar = w * amplitudes[2*(s*size_ampli+l%size_ampli)] +
	      (1-w)*amplitudes[2*(s*size_ampli + (l+1)%size_ampli)];
	    ai = w * amplitudes[2*(s*size_ampli+l%size_ampli)+1] +
	      (1-w)*amplitudes[2*(s*size_ampli + (l+1)%size_ampli)+1];
	  }
	  if (r > 0.001) {
	    // FLOAT han_r = sqrt((float)(2.0/((FLOAT)M_PI*k*r)))*cos(k*r - (FLOAT)M_PI/4.0);
	    // FLOAT han_i = sqrt((float)(2.0/((FLOAT)M_PI*k*r)))*sin(k*r - (FLOAT)M_PI/4.0);

	    uint ind = floor(k*r/0.025);
	     FLOAT coef = k*r/0.025 - ind;
	    if (ind >= 99999) {
	      ind = 0;
	      coef = 0;
	    }
#ifdef PROJECTED_GRID
	    if (s < nb_sources_input && ind < 10) {
	      ind = 10;
	      coef = 0;
	    }
#endif
	    FLOAT han_r = //hankel_r_tab[ind];
		    (1-coef)*hankel_r_tab[ind] + coef*hankel_r_tab[ind+1];
	    FLOAT han_i = //hankel_i_tab[ind];
		    (1-coef)*hankel_i_tab[ind] + coef*hankel_i_tab[ind+1];
	    FLOAT tmpr = han_r*ar - han_i*ai;
	    FLOAT tmpi = han_r*ai + han_i*ar;
	    tmpr *= damp;
	    tmpi *= damp;
	    hr += tmpr;
	    hi += tmpi;
	    // dispxr += tmpi*kx;
	    // dispxi -= tmpr*kx;
	    // dispyr += tmpi*ky;
	    // dispyi -= tmpr*ky;
	  }
	}
	//	}
      }
	
    }
    hr *= mod;
    hi *= mod;
    heights[i] += hr*cos(-omega*time) - hi*sin(-omega*time);
#ifdef PLOT_RESULT
    heights[nb_cols*nb_rows + i] = sqrt(hr*hr +  hi*hi);
#endif
    // displacement[2*i] += dispxr*cos(-omega*time) - dispxi*sin(-omega*time);
    // displacement[2*i+1] += dispyr*cos(-omega*time) - dispyi*sin(-omega*time);
    
  }
}

__global__

void addHeightM(FLOAT *heights, FLOAT *displacement, FLOAT *amplitudes, FLOAT *indexes,
		FLOAT *positions, 
		bool *is_active, FLOAT *positions_grid, FLOAT *sizes,
		uint nb_sources, uint nb_sources_input,
		FLOAT k, FLOAT omega, FLOAT vel,
		FLOAT time, FLOAT dt, uint nb_rows, uint nb_cols,
		int size_ampli, int ampli_step, FLOAT damping, FLOAT scale,
		bool show_input, bool show_scattered,
		FLOAT *hankel_r_tab, FLOAT *hankel_i_tab ) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (uint i = index; i < nb_rows*nb_cols; i += stride) {
    FLOAT wl = 2*M_PI/k;
#ifdef PROJECTED_GRID
    if (sizes[i] >= wl/2.0) {
      return;
    }
    FLOAT x = (1+positions_grid[2*i])*scale/2.0;
    FLOAT y = (1+positions_grid[2*i+1])*scale/2.0;
    // FLOAT dispxr = 0, dispyr = 0;
    // FLOAT dispxi = 0, dispyi = 0;
      FLOAT mod = 1;
      if (sizes[i] > wl/4.0) {
	mod = 1 - (sizes[i] - wl/4.0)/(wl/4.0);
      }
#else
      FLOAT x = scale/(FLOAT)nb_rows*(i/nb_cols);
      FLOAT y = scale/(FLOAT)nb_cols*(i%nb_cols);
      FLOAT mod = 1;
#endif
      FLOAT hr = 0, hi = 0;
      uint start = 0, end =  nb_sources + nb_sources_input;
      if (!show_input) {
	start = nb_sources_input;
      }
      if (!show_scattered) {
	end = nb_sources_input;
      }
      for (uint s = start; s < end; ++s) {
	
      while (!is_active[s] && s < end) {
	++s;
      }
      if (s >= end) {
	break;
      }
      int t_cur = floor((time)/((FLOAT)ampli_step*dt));
      FLOAT rmax = log(0.02)/(-damping*k*k);
      FLOAT time_lim = rmax/vel;
      int lim = 2*time_lim / (ampli_step*dt) + 1;
      int tb = t_cur-lim;
      if (tb < 0) {
	tb = 0;
      }
      for (int past_t = tb; past_t <= t_cur; ++past_t) {
	FLOAT rx = x - positions[2*(s*size_ampli + past_t)];
	FLOAT ry = y - positions[2*(s*size_ampli + past_t)+1];
	FLOAT r = sqrt((float)(rx*rx + ry*ry));
	FLOAT damp = exp(-damping*k*k*r);
	if (damp > 0.02) { 
	  FLOAT ret = r/vel;
	  int l = floor((time-ret)/((FLOAT)ampli_step*dt));
	  if (time-ret > 0) {
 	  
	    FLOAT ar = 0, ai = 0;
	     if (l >= past_t - 1 || l <= past_t +1) {
	       int pt = past_t;
	       //for (int pt = past_t - 1; pt <= past_t + 1; ++pt) {
	      FLOAT t = time-ret;
	      FLOAT w = 0;
	      FLOAT tl = pt*ampli_step*dt;
	      FLOAT tl_prev = (pt-1)*ampli_step*dt, tl_next = (pt+1)*ampli_step*dt;
	      if (t < tl_prev || t > tl_next) {
		w = 0;
	      } else if (t < tl) {
		w = (t - tl_prev)/(dt*(FLOAT)ampli_step);
	      } else {
		w = (tl_next - t)/(dt*(FLOAT)ampli_step);
	      }
	      ar = w * amplitudes[2*(s*size_ampli + pt)];
	      ai = w * amplitudes[2*(s*size_ampli + pt) + 1];
	    
	      uint ind = floor(k*r/0.025);
	      FLOAT coef = k*r/0.025 - ind;
	      if (ind >= 99999) {
		ind = 0;
		coef = 0;
	      }
#ifdef PROJECTED_GRID
	      if (s < nb_sources_input && ind < 10) {
		ind = 10;
		coef = 0;
	      }
#endif
	      FLOAT han_r = (1-coef)*hankel_r_tab[ind] + coef*hankel_r_tab[ind+1];
	      //hankel_r_tab[ind];
	      FLOAT han_i = (1-coef)*hankel_i_tab[ind] + coef*hankel_i_tab[ind+1];
	      //hankel_i_tab[ind];
	      FLOAT tmpr = han_r*ar - han_i*ai;
	      FLOAT tmpi = han_r*ai + han_i*ar;
	      tmpr *= damp;
	      tmpi *= damp;
	      hr += tmpr;
	      hi += tmpi;
   
	    }
	  }
	}
      }
    }
    hr *= mod;
    hi *= mod;
    heights[i] += hr*cos(-omega*time) - hi*sin(-omega*time);
#ifdef PLOT_RESULT
    heights[nb_cols*nb_rows + i] = sqrt(hr*hr +  hi*hi);
#endif
    // displacement[2*i] += dispxr*cos(-omega*time) - dispxi*sin(-omega*time);
    // displacement[2*i+1] += dispyr*cos(-omega*time) - dispyi*sin(-omega*time);
  }
}
      

__global__
void addHeight0(FLOAT *heights, FLOAT *displacement, FLOAT *amplitudes,
		      FLOAT *indexes, FLOAT *positions,
		      FLOAT *positions_grid, FLOAT *sizes,
		      uint nb_sources, uint nb_sources_input, FLOAT k, FLOAT omega, FLOAT vel,
		      uint nb_rows, uint nb_cols, FLOAT damping,
		      FLOAT scale, bool show_input, bool show_scattered,
		      FLOAT *hankel_r_tab, FLOAT *hankel_i_tab) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (uint i = index; i < nb_rows*nb_cols; i += stride) {
    FLOAT wl = 2.0*M_PI/k;
    FLOAT mod = 1;
#ifdef PROJECTED_GRID
    if (sizes[i] >= wl/2.0) {
      return;
    }
    if (sizes[i] > wl/4.0) {
      mod = 1 - (sizes[i] - wl/4.0)/(wl/4.0);
    }
    FLOAT x = (1+positions_grid[2*i])*scale/2.0;
    FLOAT y = (1+positions_grid[2*i+1])*scale/2.0;
#else
    FLOAT x = scale/(FLOAT)nb_rows*(i/nb_cols);
    FLOAT y = scale/(FLOAT)nb_cols*(i%nb_cols);
 #endif
     uint start = 0, end = nb_sources + nb_sources_input;
      if (!show_input) {
	start = nb_sources_input;
      }
      if (!show_scattered) {
	end = nb_sources + nb_sources_input;
      }
      for (uint s = start; s < end; ++s) {
      FLOAT hr = 0, hi = 0;
      FLOAT dispxr = 0, dispyr = 0;
      FLOAT dispxi = 0, dispyi = 0;
      if (indexes[s] == 1) {
	FLOAT dx = positions[2*s];
	FLOAT dy = positions[2*s + 1];
	FLOAT kx = k*(x*dx + y*dy);
	FLOAT ar = amplitudes[2*s];
	FLOAT ai = amplitudes[2*s + 1];
	hr = ar*cos(kx) - ai*sin(kx);
	hi = ai*cos(kx) + ar*sin(kx);
	hr *= mod;
	hi *= mod;
	dispxr = hi*dx;
	dispxi = -hr*dx;
	dispyr = hi*dy;
	dispyi = -hr*dy;
      } else {
	FLOAT rx = x - positions[2*s];
	FLOAT ry = y - positions[2*s+1];
      
	FLOAT r = sqrt((float)(rx*rx + ry*ry));
	FLOAT kx = rx/r;
	FLOAT ky = ry/r;
	FLOAT ar = amplitudes[2*s], ai = amplitudes[2*s+1];
     
	FLOAT damp = exp(-damping*k*k*r);
	//	  if (r > 0.0001) {
	// FLOAT han_r = sqrt((float)(2.0/((FLOAT)M_PI*k*r)))*cos(k*r - (FLOAT)M_PI/4.0);
	// FLOAT han_i = sqrt((float)(2.0/((FLOAT)M_PI*k*r)))*sin(k*r - (FLOAT)M_PI/4.0);

	uint ind = floor(k*r/0.025);
	FLOAT coef = k*r/0.025 - ind;
	if (ind >= 999999) {
	  ind = 0;
	  coef = 0;
	}
	FLOAT han_r = (1-coef)*hankel_r_tab[ind] + coef*hankel_r_tab[ind+1];
	FLOAT han_i = (1-coef)*hankel_i_tab[ind] + coef*hankel_i_tab[ind+1];

	hr = han_r*ar - han_i*ai;
	hi = han_r*ai + han_i*ar;
	if (k*sqrt(ai*ai + ar*ar) < 1) {
	  dispxr = hi*kx;
	  dispxi = -hr*kx;
	  dispyr = hi*ky;
	  dispyi = -hr*ky;
	}
	hr *= damp;
	hi *= damp;
	hr *= mod;
	hi *= mod;
	//  }
	 
	 
      }
      	   
#ifdef PLOT_RESULT
      if (s < nb_sources_input) {
	heights[2*i] += hr;
	heights[2*i+1] += hi;
      } else {
	heights[2*nb_rows*nb_cols + 2*i] += hr;
	heights[2*nb_rows*nb_cols + 2*i+1] += hi;
      }
      displacement[4*i] = 0;
      displacement[4*i+1] = 0;
      displacement[4*i+2] = 0;
      displacement[4*i+3] = 0;
#else
      heights[2*i] += hr;
      heights[2*i+1] += hi;
      displacement[4*i] += 0.5*dispxr;
      displacement[4*i+1] += 0.5*dispxi;
      displacement[4*i+2] += 0.5*dispyr;
      displacement[4*i+3] += 0.5*dispyi;
#endif
    }
  }
}

 
__global__
void updateHeight(FLOAT * heights, FLOAT* time_height,
		  FLOAT * displacement, FLOAT* time_displacement,
		  FLOAT omega, uint n_nodes, FLOAT time) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (uint i = index; i < n_nodes; i += stride) {
    FLOAT hr = heights[2*i];
    FLOAT hi = heights[2*i+1];
    FLOAT dispxr = displacement[4*i];
    FLOAT dispxi = displacement[4*i+1];
    FLOAT dispyr = displacement[4*i+2];
    FLOAT dispyi = displacement[4*i+3];
#ifdef PLOT_RESULT
    hr += heights[2*n_nodes + 2*i];
    hi += heights[2*n_nodes + 2*i+1];
#endif
    time_height[i] += hr*cos(-omega*time) - hi*sin(-omega*time);
    time_displacement[2*i] += dispxr*cos(-omega*time) - dispxi*sin(-omega*time);
    time_displacement[2*i+1] += dispyr*cos(-omega*time) - dispyi*sin(-omega*time);
  }
}


CudaWaterSurface::CudaWaterSurface() {
  nb_wl = 0;
}

CudaWaterSurface::~CudaWaterSurface() {}

void CudaWaterSurface::clear() {
#ifdef INTERACTIVE_
  hipFree(heights);
#endif
  for (uint w = 0; w < nb_wl; ++w) {
    hipFree(amplitudes[w]);
    hipFree(indexes[w]);
    hipFree(positions[w]);
#ifdef INTERACTIVE_
    hipFree(is_active[w]);
#else
    hipFree(heights[w]);
#endif
  }
#ifdef PROJECTED_GRID
  hipFree(positions_grid);
  hipFree(sizes);
#endif
  hipFree(hankel_r_tab);
  hipFree(hankel_i_tab);
  INFO("CLEAR CUDA");
}

 
#ifndef INTERACTIVE_

void CudaWaterSurface::init(uint nw) {
  nb_wl = nw;
  amplitudes = std::vector<FLOAT*>(nb_wl);
  indexes = std::vector<FLOAT*>(nb_wl);
  positions = std::vector<FLOAT*>(nb_wl);
  nb_sources = std::vector<uint>(nb_wl);
  nb_sources_input = std::vector<uint>(nb_wl);
  wave_lenghts = std::vector<FLOAT>(nb_wl);
  heights = std::vector<FLOAT*>(nb_wl);
  displacement = std::vector<FLOAT*>(nb_wl);

  hipMallocManaged(&time_heights, n_cols_*n_rows_*sizeof(FLOAT));
  hipMallocManaged(&time_displacement, 2*n_cols_*n_rows_*sizeof(FLOAT));
#ifdef PROJECTED_GRID
  hipMallocManaged(&positions_grid, 2*n_cols_*n_rows_*sizeof(FLOAT));
  hipMallocManaged(&sizes, n_cols_*n_rows_*sizeof(FLOAT));
#else
  positions_grid = NULL;
  sizes = NULL;
#endif
  createTabs();
}
 

void CudaWaterSurface::allocMem(uint wl, uint ns, uint na) {
  hipMallocManaged(&displacement[wl], 4*n_cols_*n_rows_*sizeof(FLOAT));
#ifdef PLOT_RESULT
  hipMallocManaged(&heights[wl], 4*n_cols_*n_rows_*sizeof(FLOAT));
  for (uint i = 0; i < 4*n_rows_*n_cols_; ++i) {
#else
  hipMallocManaged(&heights[wl], 2*n_cols_*n_rows_*sizeof(FLOAT));
  for (uint i = 0; i < 2*n_rows_*n_cols_; ++i) {
#endif
    heights[wl][i] = 0;
		    
  }
  for (uint i = 0; i < 4*n_rows_*n_cols_; ++i) {
    displacement[wl][i] = 0;
  }

  hipMallocManaged(&amplitudes[wl], 2*(ns+na)*sizeof(FLOAT));
  hipMallocManaged(&indexes[wl], (ns+na)*sizeof(FLOAT));
  hipMallocManaged(&positions[wl], 2*(ns+na)*sizeof(FLOAT));
  nb_sources[wl] = ns;
  nb_sources_input[wl] = na;
}

void CudaWaterSurface::setHeight(int nb_in_waves) {
  
  for (uint w = 0; w < nb_wl; ++w) {
    
#ifdef PLOT_RESULT
    uint nb_f = 4*n_rows_*n_cols_;
#else
    uint nb_f = 2*n_rows_*n_cols_;
#endif  
    for (uint i = 0; i < nb_f; ++i) {
      heights[w][i] = 0;
    }
    for (uint i = 0; i < 4*n_rows_*n_cols_; ++i) {
      displacement[w][i] = 0;
    }
    FLOAT wl = wave_lenghts[w];
    FLOAT k = 2*M_PI/wl;
    FLOAT omega = angular_vel(k);
    FLOAT v = velocity(k);
    int blockSize = 256;
    int numBlocks = ( n_rows_*n_cols_ + blockSize - 1) / blockSize;

    uint ns = nb_sources[w];
    uint na = nb_sources_input[w];
    addHeight0<<<numBlocks,blockSize>>>
      (heights[w], displacement[w], amplitudes[w], indexes[w], positions[w], 
       positions_grid, sizes, ns, na,
       k, omega, v, n_rows_, n_cols_,
       damping_, scale_,
       ui_parameters::show_in_field , ui_parameters::show_scattered_field,
       hankel_r_tab, hankel_i_tab);
    hipDeviceSynchronize();
  }
  
}
 
void CudaWaterSurface::setTimeHeight(int time) {
  for (uint i = 0; i < n_rows_*n_cols_; ++i) {
    time_heights[i] = 0;
  }
  for (uint i = 0; i < 4*n_rows_*n_cols_; ++i) {
    time_displacement[i] = 0;
  }

  for (uint w = 0; w < nb_wl; ++w) {
    FLOAT wl = wave_lenghts[w];
    FLOAT k = 2*M_PI/wl;
    FLOAT omega = angular_vel(k);
    int blockSize = 256;
    int numBlocks = ( n_rows_*n_cols_ + blockSize - 1) / blockSize;
      
    updateHeight<<<numBlocks,blockSize>>>(heights[w], time_heights,
					  displacement[w], time_displacement,
					  omega, n_cols_*n_rows_, time*dt_);
    hipDeviceSynchronize();
  }
}

#else

void CudaWaterSurface::init(uint nw) {
  nb_wl = nw;
  amplitudes = std::vector<FLOAT*>(nb_wl);
  indexes = std::vector<FLOAT*>(nb_wl);
  positions = std::vector<FLOAT*>(nb_wl);
  nb_sources = std::vector<uint>(nb_wl);
  nb_sources_input = std::vector<uint>(nb_wl);
  wave_lenghts = std::vector<FLOAT>(nb_wl);
  is_active = std::vector<bool *>(nb_wl);

#ifdef PLOT_RESULT
  uint h_size = 2*n_cols_*n_rows_;
#else
  uint h_size = n_cols_*n_rows_;
#endif
 hipMallocManaged(&heights, h_size*sizeof(FLOAT));
 for (uint i = 0; i < h_size; ++i) {
   heights[i] = 0;
 }
 hipMallocManaged(&displacement, 2*n_cols_*n_rows_*sizeof(FLOAT));
 for (uint i = 0; i < 2*n_rows_*n_cols_; ++i) {
   displacement[i] = 0;
 }
#ifdef PROJECTED_GRID
 hipMallocManaged(&positions_grid, 2*n_cols_*n_rows_*sizeof(FLOAT));
 hipMallocManaged(&sizes, n_cols_*n_rows_*sizeof(FLOAT));
#else
  positions_grid = NULL;
  sizes = NULL;
#endif
  createTabs();
  hipMallocManaged(&buffer, nb_profil*sizeof(FLOAT));
}

void CudaWaterSurface::allocMem(uint wl, uint ns, uint na) {
  hipError_t alloc_ok = hipMallocManaged(&amplitudes[wl], 2*(ns+na)*size_tmp*sizeof(FLOAT));
  hipMallocManaged(&indexes[wl], (ns+na)*sizeof(FLOAT));
  hipMallocManaged(&is_active[wl], (ns+na)*sizeof(bool));
  hipMallocManaged(&positions[wl], 2*(ns+na)*sizeof(FLOAT));
  nb_sources[wl] = ns;
  nb_sources_input[wl] = na;
}


void CudaWaterSurface::setHeight(int time) {
  for (uint i = 0; i < n_rows_*n_cols_; ++i) {
    heights[i] = 0;
  }
  for (uint i = 0; i < 2*n_rows_*n_cols_; ++i) {
    displacement[i] = 0;
  }
	
  for (uint w = 0; w < nb_wl; ++w) {
    FLOAT wl = wave_lenghts[w];
    FLOAT k = 2*M_PI/wl;
    FLOAT omega = angular_vel(k);
    int blockSize = 64;
    int numBlocks = ( n_rows_*n_cols_ + blockSize - 1) / blockSize;
    FLOAT v = velocity(k);

    addHeight<<<numBlocks,blockSize>>>
      (heights, displacement, amplitudes[w], indexes[w],
       positions[w],  is_active[w],
       positions_grid, sizes, 
       nb_sources[w], nb_sources_input[w],
       k, omega, v, time*dt_, dt_, n_rows_, n_cols_,
       size_tmp, ampli_steps[w], damping_, scale_,
       ui_parameters::show_in_field, ui_parameters::show_scattered_field,
       hankel_r_tab, hankel_i_tab);
     hipDeviceSynchronize();
    addHeightM<<<numBlocks,blockSize>>>
      (heights, displacement, amplitudes_m[w], indexes_m[w],
       positions_m[w], is_active_m[w],
       positions_grid, sizes,
       nb_sources_m[w], nb_sources_input_m[w],
       k, omega, v, time*dt_, dt_, n_rows_, n_cols_,
       size_tmp, ampli_steps[w], damping_, scale_,
       ui_parameters::show_in_field, ui_parameters::show_scattered_field,
       hankel_r_tab, hankel_i_tab);
    hipDeviceSynchronize();
  }

}


void CudaWaterSurface::initM(uint nw) {
  amplitudes_m = std::vector<FLOAT*>(nb_wl);
  indexes_m = std::vector<FLOAT*>(nb_wl);
  positions_m = std::vector<FLOAT*>(nb_wl);
  nb_sources_m = std::vector<uint>(nb_wl);
  nb_sources_input_m = std::vector<uint>(nb_wl);
  is_active_m = std::vector<bool *>(nb_wl);
}

void CudaWaterSurface::allocMemM(uint wl, uint ns, uint na) {
  hipError_t alloc_ok = hipMallocManaged(&amplitudes_m[wl], 2*(ns+na)*size_tmp*sizeof(FLOAT));
  hipMallocManaged(&indexes_m[wl], (ns+na)*sizeof(FLOAT));
  hipMallocManaged(&is_active_m[wl], (ns+na)*sizeof(bool));
  hipMallocManaged(&positions_m[wl], 2*(ns+na)*size_tmp*sizeof(FLOAT));
  nb_sources_m[wl] = ns;
  nb_sources_input_m[wl] = na;
}


      

      
#endif


void CudaWaterSurface::createTabs() {
  hipMallocManaged(&hankel_r_tab, nb_profil*sizeof(FLOAT));
  hipMallocManaged(&hankel_i_tab, nb_profil*sizeof(FLOAT));

  for (uint i = 0; i < settings::nb_profil; ++i) {
    COMPLEX h = settings::hankel_tab[i];
    hankel_r_tab[i] = real(h);
    hankel_i_tab[i] = imag(h);
  }
  for (uint i = 0; i < 1; ++i) {
    hankel_r_tab[i] = hankel_r_tab[1];
    hankel_i_tab[i] = hankel_i_tab[1];
  }
}
			
